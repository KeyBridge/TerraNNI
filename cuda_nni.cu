#include "hip/hip_runtime.h"
// -*- mode: c++; tab-width: 4; indent-tabs-mode: t; eval: (progn (c-set-style "stroustrup") (c-set-offset 'innamespace 0)); -*-
// vi:set ts=4 sts=4 sw=4 noet :

/*
 * TerraNNI: Natural neighbor interpolation in 2D and 3D for large datasets
 * Copyright (C) 2010, 2011: Pankaj K. Agarwal, Alex Beutel & Thomas Mølhave
 *
 * This file is part of TerraNNI.
 *
 * TerraNNI is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * TerraNNI is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with TerraNNI.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "nni.h"
#include "cuda_nni.h"

using namespace std;
//using namespace nni;
using namespace cuda_nni;

#include <iostream>
#include <cassert>

#include <cmath>
#include <map>
#include <algorithm>
#include <string.h>

#include </usr/include/GL/glew.h>

//#include <hip/hip_runtime_api.h>
//#include <helper_cuda_gl.h>

//#include "cutil_inline.h"
//#include <cutil_gl_inline.h>

//#include <cudaGL.h>

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
//#include <cutil_gl_error.h>

#include <cstdlib>

#include <GL/glut.h>
#include <GL/glext.h>
//#include <GL/gl.h>
//#include <GL/glu.h>
#include <GL/glx.h>


#define GL_ERROR2() CheckGLError2(__FILE__, __LINE__)
bool CheckGLError2(char* acSourceFile, int iLine)
{
	GLenum eErr;
	bool bError = false;
	eErr = glGetError();
	while (eErr != GL_NO_ERROR) {
		fprintf(stderr, "OpenGL: %s, errno %d, source file %s, source line %d\n", gluErrorString(eErr), eErr, acSourceFile, iLine);
		bError = true;
		eErr = glGetError();
	}
	return bError;
}

size_t checkMemory() {

	size_t theFree, theTotal;
	hipError_t res = hipMemGetInfo( &theFree, &theTotal );
	if(res != hipSuccess)
		cerr << "hipMemGetInfo Failed\n";
	cerr << "Memory: " << theFree << " :: " << theTotal << "\n";

	return theFree;

}

texture<float4, 2, hipReadModeElementType> siteTex;
texture<float4, 2, hipReadModeElementType> queryTex;

/* -------------------------------
Offscreen rendering
------------------------------- */

GLuint vboNum, vboDenom;
struct hipGraphicsResource *vbo_res_num;
struct hipGraphicsResource *vbo_res_denom;
bool vboInitialized = false;

struct hipGraphicsResource* siteBuffer_CUDA;
struct hipGraphicsResource* queryBuffer_CUDA;

struct hipGraphicsResource* siteBuffer_CUDA_depth;

hipGraphicsResource ** allSiteBuffers_Color;
hipGraphicsResource ** allSiteBuffers_Depth;

// Initialize vertex buffer objects (VBOs) to store the numerator and
// denominator during NNI computation
void initVBO(int w_width, int w_height) {
	fprintf(stderr, "VBO Initialized: %d x %d\n", w_width, w_height);
	vboInitialized = true;

	unsigned int size = w_width * w_height * sizeof(int);	
	glGenBuffers(1, &vboNum);
	glBindBuffer(GL_ARRAY_BUFFER, vboNum);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	hipGraphicsGLRegisterBuffer(&vbo_res_num, vboNum, cudaGraphicsMapFlagsNone);

	glGenBuffers(1, &vboDenom);
	glBindBuffer(GL_ARRAY_BUFFER, vboDenom);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	hipGraphicsGLRegisterBuffer(&vbo_res_denom, vboDenom, cudaGraphicsMapFlagsNone);
}

int * rb_voronois_indices;
int rb_query_index;

void cuda_nni::cudaRegisterAllBuffers(int tC, int * rb_v, int rb_q) {

	rb_voronois_indices=rb_v; 
	rb_query_index=rb_q; 

	/*

	//unsigned int theFree, theTotal,theFree2;
	size_t theFree, theTotal,theFree2;
	hipError_t res = hipMemGetInfo( &theFree, &theTotal );

	//hipGraphicsResource * allSiteBuffers_Color[tC];
	allSiteBuffers_Color = new hipGraphicsResource*[tC];

	for(int i = 0; i < tC; i++) {
		if(true) continue;
		//if(i == 1) continue;

		fprintf(stderr, "Register Voronoi render buffer %d\n", rb_voronois[i]);

		res = hipMemGetInfo( &theFree, &theTotal );
		if(res != hipSuccess)
			fprintf(stderr, "hipMemGetInfo FAILED\n");
		fprintf(stderr, "Memory: %u :: %u\n", theFree, theTotal);


		//cutilSafeCall(hipGraphicsGLRegisterImage(&allSiteBuffers_Color[i], rb_voronois[i], GL_RENDERBUFFER, cudaGraphicsMapFlagsReadOnly));
		hipError_t reg_res = hipGraphicsGLRegisterImage(&allSiteBuffers_Color[i], rb_voronois[i], GL_RENDERBUFFER, cudaGraphicsMapFlagsReadOnly);
		if(reg_res != hipSuccess) {
			cout << "Error registering image\n\n";
		}


		//hipFreeArray(allSiteBuffers_Color[i]);
		//cutilSafeCall(cudaGraphicsGLUnregisterImage(allSiteBuffers_Color[i]));
		hipError_t res2 = hipGraphicsUnregisterResource(allSiteBuffers_Color[i]);
		if(res2 != hipSuccess) {
			fprintf(stderr, "*******error unregistering resource*******\n\n");
		}

		res = hipMemGetInfo( &theFree2, &theTotal );
		if(res != hipSuccess)
			fprintf(stderr, "hipMemGetInfo FAILED\n");
		fprintf(stderr, "Memory: %u :: %u\n", theFree2, theTotal);
		fprintf(stderr, "Memory used: %d\n\n", (theFree-theFree2));
	}

	fprintf(stderr, "Register Query render buffer %d\n", rb_query);
	res = hipMemGetInfo( &theFree, &theTotal );
	if(res != hipSuccess)
		fprintf(stderr, "hipMemGetInfo FAILED\n");
	fprintf(stderr, "Memory: %u :: %u\n", theFree, theTotal);

	cutilSafeCall(hipGraphicsGLRegisterImage(&queryBuffer_CUDA, rb_query, GL_RENDERBUFFER, cudaGraphicsMapFlagsReadOnly));

	res = hipMemGetInfo( &theFree, &theTotal );
	if(res != hipSuccess)
		fprintf(stderr, "hipMemGetInfo FAILED\n");
	fprintf(stderr, "Memory: %u :: %u\n", theFree, theTotal);
     */



	cudaSwitchVoronoiRB(0);
	//cudaSwitchVoronoiRB(rb_voronois[0]);
}


// Switch to renderbuffer at index rbIndex
void cuda_nni::cudaSwitchVoronoiRB(int rbIndex) {
	//siteBuffer_CUDA = allSiteBuffers_Color[rbIndex];


	//Unregister previous renderbuffers if they exist
	hipError_t res2 = hipGraphicsUnregisterResource(siteBuffer_CUDA);
	if(res2 != hipSuccess) {
		fprintf(stderr, "*******error unregistering resource*******\n\n");
	}

	res2 = hipGraphicsUnregisterResource(queryBuffer_CUDA);
	if(res2 != hipSuccess) {
		fprintf(stderr, "*******error unregistering query resource*******\n\n");
	}


	cout << "Switch to renderbuffer " << rb_voronois_indices[rbIndex] << "\n";

	size_t theFree, theFree2;

	theFree = checkMemory();

	hipError_t reg_res = hipGraphicsGLRegisterImage(&siteBuffer_CUDA, rb_voronois_indices[rbIndex], GL_RENDERBUFFER, cudaGraphicsMapFlagsReadOnly);
	if(reg_res != hipSuccess) {
		cout << "Error registering image\n\n";
	}

	theFree2 = checkMemory();
	cerr << "Memory used: " << (theFree-theFree2) << "\n";



	fprintf(stderr, "Register Query render buffer %d\n", rb_query_index);
	theFree = checkMemory();
	hipGraphicsGLRegisterImage(&queryBuffer_CUDA, rb_query_index, GL_RENDERBUFFER, cudaGraphicsMapFlagsReadOnly);
	theFree2 = checkMemory();
	cerr << "Memory used: " << (theFree-theFree2) << "\n";

}


void init() {
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glEnable(GL_DEPTH_TEST);
}

void reshape(int w, int h) {
	glViewport(0,0, (GLsizei) w, (GLsizei) h);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(-5.0, 20.0, -5.0, 20.0, -50.0, 50.0);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
}

void fakeDisplayFunc() { }


// Start GLUT
void startWithGLUT() {
	int argc =0;	
	char * argv = "";

	glutInit ( &argc, &argv );
	glutInitDisplayMode ( GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
	glutInitWindowSize ( 800, 800 );
	glutCreateWindow ( "TerraNNI" );
	init();
	glutDisplayFunc ( fakeDisplayFunc );
	glutReshapeFunc ( reshape );
	
	glewInit();

}

// Attempt to not create a window (without GLUT) for true offscreen rendering
// Has never quite worked
void startOffscreen2() {	
	Display *display = XOpenDisplay(0);
	XVisualInfo *vinfo;
	int attrList[20];
	int indx=0;
	GLXContext util_glctx;

//   Colormap cmap;
//   XSetWindowAttributes swa;
//   Window win;
//   XEvent event;

	if(!display) exit (1);
	
	attrList[indx] = GLX_USE_GL; indx++;
	attrList[indx] = GLX_RGBA; indx++;
	attrList[indx] = GLX_DEPTH_SIZE; indx++;
	attrList[indx] = 8; indx++;
	attrList[indx] = GLX_RGBA; indx++;
	attrList[indx] = GLX_RED_SIZE; indx++;
	attrList[indx] = 8; indx++;
	attrList[indx] = GLX_GREEN_SIZE; indx++;
	attrList[indx] = 8; indx++;
	attrList[indx] = GLX_BLUE_SIZE; indx++;
	attrList[indx] = 8; indx++;
	attrList[indx] = None;

	vinfo = glXChooseVisual(display, DefaultScreen(display), attrList);
	if (vinfo == NULL) {
		printf ("ERROR: Can't open window\n");
		exit (1);
	}

	Pixmap pixmap = XCreatePixmap(display, DefaultRootWindow(display), 1,1,vinfo->depth);
	GLXPixmap glxpix = glXCreateGLXPixmap(display, vinfo, pixmap);

	util_glctx = glXCreateContext(display, vinfo, NULL, False);
	if (util_glctx == NULL) {
		printf("glXCreateContext failed \n");
		return;
	}

	if (!glXMakeCurrent(display, glxpix, util_glctx)) {
		printf("glXMakeCurrent failed \n");
		exit(1);
	}
	
	glewInit();

	if(glXGetCurrentContext() == NULL) {
		fprintf(stderr, "Uh oh, no context is currently set...\n");
	}
	
}


/* -------------------------------
End offscreen rendering
------------------------------- */

/* -------------------------------
CUDA functions
------------------------------- */


// Set all values to zero in the array
// Used for zeroing the VBOs
__global__ void zeroValues(int1 * arrayPoint, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < N) arrayPoint[i].x = 0;
}




__device__ void cudaCheckBits(uint t, // value in query buffer (representing the bitwise-OR of rendered queries 
		int1 * denom,  
		int1 * numer, 
		uint nearestPointVal, // value in color buffer, which is the value (usually elevation of the nearest neighbor to the pixel
		int x_pixel, 
		int y_pixel, 
		int ncols, 
		int nrows, 
		int blockWidth, 
		int scaling, 
		int pixel_offset, 
		int curRound, 
		int spacingFactor) 
{

	// Only continue if both a query cone AND a input point covered the pixel
    if(t != 0 && nearestPointVal > 0) {
		int rBlockWidth = blockWidth * spacingFactor;
		int nrounds = spacingFactor*spacingFactor;

		// This code assumes we only have 32 bits in the color buffer
		// Loop through each bit in the depth buffer
		for(int ti = 0; ti < 32; ti++) {
			// If the bit is set to one then a query covered it
			if( (t >> ti) & 1 == 1) {
				
				// Find the query that was rendered
				int offset_i = (int)(scaling * (0.5 + ((ti*nrounds + curRound) % rBlockWidth))) + pixel_offset;
				int offset_j = (int)(scaling * (0.5 + ((ti*nrounds + curRound) / rBlockWidth))) + pixel_offset;

				int block_i = round((x_pixel - offset_i) / (float)(rBlockWidth * scaling));
				int block_j = round((y_pixel - offset_j) / (float)(rBlockWidth * scaling));
				
				int p_i = block_i * rBlockWidth + ((ti*nrounds + curRound) % rBlockWidth);
				int p_j = block_j * rBlockWidth + ((ti*nrounds + curRound) / rBlockWidth);
				
				int whole_index = p_j * ncols + p_i;
				
				// Increment the numerator and denominator for that query
				if(whole_index < ncols * nrows && whole_index >= 0) {
					atomicAdd(&(numer[whole_index].x),nearestPointVal );
					atomicAdd(&(denom[whole_index].x),1);
				}		
			}
		}
    }

}



// General function to perform Buffer Analysis
__global__ void calcNNI(int1 * numer, int1 * denom, int width, int height, int scaling, int blockWidth, int pixel_offset, int curRound, int spacingFactor) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	// (x,y) is a pixel location in the buffers
    // confirm that the pixel is in bounds	
	if(x < ((width*scaling)+2*pixel_offset) && y < ((height*scaling)+2*pixel_offset)) {

		// Read RGBA values from the two buffers
		float4 voronoi = tex2D(siteTex, x, y);
		float4 query = tex2D(queryTex, x, y);

		// Combine the RGBA values
		int vx = *((int *)(&(voronoi.x)));
		int vy = *((int *)(&(voronoi.y)));
		int vz = *((int *)(&(voronoi.z)));
		int vw = *((int *)(&(voronoi.w)));
		uint v = vx | (vy << 8) | (vz << 16) | (vw << 24); // do i need to cast to uint before the shfit -amb79

		int tx = *((int *)(&(query.x)));
		int ty = *((int *)(&(query.y)));
		int tz = *((int *)(&(query.z)));
		int tw = *((int *)(&(query.w)));
		uint t = tx | (ty << 8) | (tz << 16) | (tw << 24);  // do i need to cast to uint before the shfit -amb79

		// Use read values
		cudaCheckBits(t,denom, numer,v,x,y, width,height, blockWidth, scaling, pixel_offset, curRound, spacingFactor);
    }

}

// Given the numerator and denominator, compute the final values
__global__ void divide(int1 * numer, int1 * denom, int width, int height, int zMult) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(y < height && x < width) {	
		// If the query did not reach any colored pixels then set the result to NODATA
		if(denom[y*width+x].x == 0)	
			denom[y*width+x].x = (-9999 * zMult);
		else 
			denom[y*width + x].x = numer[y*width+x].x/denom[y*width+x].x;
	}
	
}


/* -------------------------------
End CUDA functions
------------------------------- */


int1 *dptrNum;
int1 *dptrDenom;


// Set up a run in CUDA
void cuda_nni::cudaSetup(int nrows, int ncols) {

	size_t num_bytes; 
	size_t num_bytes2; 

	cerr << "Map VBO resuorces\n";
	hipGraphicsMapResources(1, &vbo_res_num, 0);
	hipGraphicsMapResources(1, &vbo_res_denom, 0);
	hipGraphicsResourceGetMappedPointer((void **)&dptrNum, &num_bytes, vbo_res_num);
	hipGraphicsResourceGetMappedPointer((void **)&dptrDenom, &num_bytes2, vbo_res_denom);

	hipGraphicsMapResources(1, &siteBuffer_CUDA, 0); 
	hipArray * siteArray;
	hipGraphicsSubResourceGetMappedArray(&siteArray, siteBuffer_CUDA, 0,0);
	hipBindTextureToArray(siteTex, siteArray);

	hipGraphicsMapResources(1, &queryBuffer_CUDA, 0); 
	hipArray * queryArray;
	hipGraphicsSubResourceGetMappedArray(&queryArray, queryBuffer_CUDA, 0,0);
	hipBindTextureToArray(queryTex, queryArray);

	// Zero VBO values
	int N = nrows * ncols;
	int threadsPerBlock = 1000;
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	fprintf(stderr, "Zero values\n");
	zeroValues<<<blocksPerGrid, threadsPerBlock>>>(dptrNum, N);
	zeroValues<<<blocksPerGrid, threadsPerBlock>>>(dptrDenom, N);
	hipDeviceSynchronize();

	hipGraphicsUnmapResources(1, &queryBuffer_CUDA, 0); 
	hipGraphicsUnmapResources(1, &siteBuffer_CUDA, 0); 
}

// Perform buffer analysis
__host__ void cudaBufferAnalysis(int nrows, int ncols, int scaling, int blockWidth, int pixel_offset, int curRound, int spacingFactor) {

	int p_height = scaling * nrows + 2 * pixel_offset;
	int p_width = scaling * ncols + 2 * pixel_offset;


	// Map renderbuffers for use
	hipGraphicsMapResources(1, &siteBuffer_CUDA, 0); 
	hipArray * siteArray;
	hipGraphicsSubResourceGetMappedArray(&siteArray, siteBuffer_CUDA, 0,0);
	hipBindTextureToArray(siteTex, siteArray);

	hipGraphicsMapResources(1, &queryBuffer_CUDA, 0); 
	hipArray * queryArray;
	hipGraphicsSubResourceGetMappedArray(&queryArray, queryBuffer_CUDA, 0,0);
	hipBindTextureToArray(queryTex, queryArray);

	//TODO: Do this value for nthreads need to be taken from CUDA?
	int nthreads = 32;
	dim3 blocks((p_width + nthreads - 1) / nthreads,(p_height + nthreads - 1) / nthreads);//(250,250);
	dim3 threads(nthreads,nthreads);//(20,20);
	calcNNI<<<blocks, threads>>>(dptrNum, dptrDenom, ncols, nrows, scaling, blockWidth, pixel_offset, curRound, spacingFactor);
	hipDeviceSynchronize();

	/*
	cutilSafeCall(hipUnbindTexture(siteTex));
	cutilSafeCall(hipUnbindTexture(queryTex));
	*/

	//cutilSafeCall(hipGraphicsUnmapResources(1, &siteArray, 0)); 
	//cutilSafeCall(hipGraphicsUnmapResources(1, &queryArray, 0)); 


	// Unmap renderbuffers
	// If this is not done then future OpenGL calls will produce unexpected results
	hipGraphicsUnmapResources(1, &queryBuffer_CUDA, 0); 
	hipGraphicsUnmapResources(1, &siteBuffer_CUDA, 0); 

}


// Complete buffer analysis
__host__ void cudaCompleteAnalysis(int nrows, int ncols, int ** res, int zMult) {

	//TODO: Do this value for nthreads need to be taken from CUDA?
	int nthreads = 32;

	// Divide the values in numerator and denominator
	dim3 blocks2((ncols + nthreads - 1) / nthreads,(nrows + nthreads - 1) / nthreads);
	dim3 threads2(nthreads,nthreads);
	divide<<<blocks2, threads2>>>(dptrNum, dptrDenom, ncols, nrows, zMult);
	hipDeviceSynchronize();

	// Unmap numerator VBO
	hipGraphicsUnmapResources(1, &vbo_res_num, 0);
	hipGraphicsUnregisterResource(vbo_res_num);
	glBindBuffer(GL_ARRAY_BUFFER_ARB, vboNum );

	// Unmap denominator VBO
	hipGraphicsUnmapResources(1, &vbo_res_denom, 0);
	hipGraphicsUnregisterResource(vbo_res_denom);
	glBindBuffer(GL_ARRAY_BUFFER_ARB, vboDenom );
	// Read values from denominator
	int * data = (int*) glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);
	
	*res = data;

}

void cuda_nni::completeAnalysis(int nrows, int ncols, int ** res, int zMult) {
	cudaCompleteAnalysis(nrows,ncols,res,zMult);
}

void cuda_nni::cleanupCUDA() {
	cerr << "Clean up CUDA\n";

	glBindBuffer(GL_ARRAY_BUFFER, vboDenom );
	glUnmapBuffer(GL_ARRAY_BUFFER);

	glDeleteBuffers(1, &vboDenom);
	glDeleteBuffers(1, &vboNum);
}

void cuda_nni::bufferAnalysis(const nni::WorkPackage& wp, int scaling, int blockWidth, 
		int pixel_offset, int curRound, int spacingFactor) { 

	cudaBufferAnalysis(wp.nrows, wp.ncols, scaling, blockWidth, pixel_offset,curRound,spacingFactor);
}

// Initialize graphics card environment and CUDA
void cuda_nni::init(const nni::Settings& s) {	
	
	// Setup OpenGL with GLUT
	startWithGLUT();
	//startOffscreen2();



	//int deviceCount = 0;



	// Set CUDA
	//cerr << "Max gflops: "<< cutGetMaxGflopsDeviceId() << "\n";
	//hipSetDevice(cutGetMaxGflopsDeviceId());
	//cudaGLSetGLDevice(cutGetMaxGflopsDeviceId());

	hipSetDevice(0);
	cudaGLSetGLDevice(0);

	hipDeviceProp_t deviceProps;
    //hipGetDeviceProperties(&deviceProps, cutGetMaxGflopsDeviceId());
    hipGetDeviceProperties(&deviceProps, 0);
	cerr << "CUDA device [" << deviceProps.name << "] has " << deviceProps.multiProcessorCount << "Multi-Processors\n";

	checkMemory();

}

void cuda_nni::setup_run(int nrows, int ncols) {
	cout << "CUDA setup run \n";
	initVBO(ncols, nrows);
}

